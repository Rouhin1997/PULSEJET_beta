#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/reduce.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/tuple.h>
#include <thrust/fill.h>
#include <kernels/defaults.h>
#include <utils/exceptions.hpp>

//--------------Harmonic summing----------------//

__global__ 
void harmonic_sum_kernel_generic(float *d_idata, float *d_odata,
				 int gulp_index, int size, int harmonic, 
				 float one_over_sqrt_harm)
{
  int Index = blockIdx.x * blockDim.x + threadIdx.x;
  if(Index<size) //This is a bug!
    {
      d_odata[gulp_index+Index] = d_idata[gulp_index+Index];
      for(int i = 1; i < harmonic; i++)
        {
          d_odata[gulp_index+Index] += d_idata[int((i*(gulp_index+Index))/harmonic+0.5)];
        }
      d_odata[gulp_index+Index] = d_odata[gulp_index+Index] * one_over_sqrt_harm;
    }
  return;
}

void device_harmonic_sum(float* d_input_array, float* d_output_array,
			 int original_size, int harmonic, 
			 unsigned int max_blocks, unsigned int max_threads)
{
  int gulps;
  int gulp_counter;
  int gulp_index = 0;
  int gulp_size;
  int blocks = 0;
  float one_over_sqrt_harm = 1.0f/sqrt((float)harmonic);
  gulps = original_size/(max_blocks*max_threads)+1;
  for(gulp_counter = 0; gulp_counter<gulps; gulp_counter++)
    {
      if(gulp_counter<gulps-1)
        {
          gulp_size = max_blocks*max_threads;
        }
      else
        {
          gulp_size = original_size - gulp_counter*max_blocks*max_threads;
        }
      blocks = (gulp_size-1)/MAX_THREADS + 1;
      harmonic_sum_kernel_generic<<<blocks,max_threads>>>(d_input_array,d_output_array,
							  gulp_index,gulp_size,harmonic,
							  one_over_sqrt_harm);
      gulp_index = gulp_index + blocks*max_threads;
    }
  ErrorChecker::check_cuda_error();
  return;
}

//------------spectrum forming--------------//

__global__ 
void power_series_kernel(hipfftComplex *d_idata,float* d_odata, int size)
{
  float* d_idata_float = (float*)d_idata;
  int Index = blockIdx.x * blockDim.x + threadIdx.x;
  if(Index<size)
    {
      d_odata[Index] = sqrtf(d_idata_float[2*Index]*d_idata_float[2*Index]
                             + d_idata_float[2*Index+1]*d_idata_float[2*Index+1]);
    }
  return;
}

__global__ void bin_interbin_series_kernel(hipfftComplex *d_idata,float* d_odata, int size)
{
  float* d_idata_float = (float*)d_idata;
  int Index = blockIdx.x * blockDim.x + threadIdx.x;
  float re_l =0.0;
  float im_l =0.0;
  if (Index>0 && Index<size) {
    re_l = d_idata_float[2*Index-2];
    im_l = d_idata_float[2*Index-1];
  }
  if(Index<size)
    {
      float re = d_idata_float[2*Index];
      float im = d_idata_float[2*Index+1];
      float ampsq = re*re+im*im;
      float ampsq_diff = 0.5*((re-re_l)*(re-re_l) +
                              (im-im_l)*(im-im_l));
      d_odata[Index] = sqrt(max(ampsq,ampsq_diff));
    }
  return;
}

void device_form_power_series(hipfftComplex* d_array_in, float* d_array_out,
			      int size, int way)
{
  int gulps;
  int gulp_counter;
  hipfftComplex* gulp_in_ptr = d_array_in;
  float* gulp_out_ptr = d_array_out;
  
  int gulp_size;
  
  gulps = (size-1)/(MAX_BLOCKS*MAX_THREADS)+1;

  for(gulp_counter = 0; gulp_counter<gulps; gulp_counter++)
    {
      if(gulp_counter<gulps-1)
        {
          gulp_size = MAX_BLOCKS*MAX_THREADS;
        }
      else
        {
          gulp_size = size - gulp_counter*MAX_BLOCKS*MAX_THREADS;
        }
      if (way==0)
        power_series_kernel<<<MAX_BLOCKS,MAX_THREADS>>>(gulp_in_ptr,gulp_out_ptr,gulp_size);
      if (way==1)
        bin_interbin_series_kernel<<<MAX_BLOCKS,MAX_THREADS>>>(gulp_in_ptr,gulp_out_ptr,gulp_size);
      gulp_in_ptr = gulp_in_ptr + MAX_BLOCKS*MAX_THREADS;
      gulp_out_ptr = gulp_out_ptr + MAX_BLOCKS*MAX_THREADS;
    }
  return;
}

//-----------------time domain resampling---------------//

__global__
void jstretch_kernel( float* d_odata, float* d_idata,
		      int start_index, int length,
		      float a, float timestep)
{
  double T = timestep*((float)length-1.0);
  double c = (float)299792458.0;

  double A = a/2.0;
  double B = -(a*T/2.0+c);

  double tobs;

  double xmax = -a*T*T/8.0;
  double dmax = (double)xmax/(double)c;


  unsigned int index = start_index + blockIdx.x*blockDim.x + threadIdx.x;

  if(index < length)
    {
      tobs = (double)index*timestep;
      double C = a*T*T/8.0 + c*tobs;
      float read_location;
      read_location = (dmax + (-B - sqrt(B*B - 4.0*A*C))/(2.0*A))/timestep;
      d_odata[index] = d_idata[(int)read_location] 
	+ (d_idata[1+(int)read_location] 
	   - d_idata[(int)read_location])
	*(read_location - (int)read_location);
    }
}

void device_resample(float * d_idata, float * d_odata,
		     unsigned int length, float a, 
		     float timestep, unsigned int block_size,
		     unsigned int max_blocks)
{
  dim3 dimBlock(block_size, 1, 1);
  int start_index;
  int gulp_length;
  start_index = 0;
  
  while(start_index < (int)length)
    {
      if(length - start_index >= max_blocks*block_size)
        {
          gulp_length = max_blocks*block_size;
        }
      else
        {
          gulp_length = length - start_index;
        }
      
      int blocks = (gulp_length - 1)/block_size + 1;
      
      dim3 dimGrid(blocks, 1, 1);
      
      jstretch_kernel<<< dimGrid, dimBlock, 0 >>>(d_odata+start_index, d_idata+start_index, start_index, gulp_length,(float)a,(float)timestep);
      start_index += gulp_length;
    }
  ErrorChecker::check_cuda_error();
}

//------------------peak finding-----------------//
//defined here as (although Thrust based) requires CUDA functors

struct greater_than_threshold : thrust::unary_function<thrust::tuple<int,float>,bool>
{
  float threshold;
  __device__ bool operator()(thrust::tuple<int,float> t) { return thrust::get<1>(t) > threshold; }
  greater_than_threshold(float thresh):threshold(thresh){}
};

int device_find_peaks(int n, int start_index, float * d_dat,
	     float thresh, int * indexes, float * snrs)
{
  using thrust::tuple;
  using thrust::counting_iterator;
  using thrust::zip_iterator;
  // Wrap the device pointer to let Thrust know                              
  thrust::device_ptr<float> dptr_dat(d_dat + start_index);
  thrust::device_vector<int> d_index(n-start_index);
  thrust::device_vector<float> d_snrs(n-start_index);
  typedef thrust::device_vector<float>::iterator snr_iterator;
  typedef thrust::device_vector<int>::iterator indices_iterator;
  thrust::counting_iterator<int> iter(start_index);
  zip_iterator<tuple<counting_iterator<int>,thrust::device_ptr<float> > > zipped_iter = make_zip_iterator(make_tuple(iter,dptr_dat));
  zip_iterator<tuple<indices_iterator,snr_iterator> > zipped_out_iter = make_zip_iterator(make_tuple(d_index.begin(),d_snrs.begin()));
  int num_copied = thrust::copy_if(zipped_iter, zipped_iter+n-start_index,zipped_out_iter,greater_than_threshold(thresh)) - zipped_out_iter;
  thrust::copy(d_index.begin(),d_index.begin()+num_copied,indexes);
  thrust::copy(d_snrs.begin(),d_snrs.begin()+num_copied,snrs);

  ErrorChecker::check_cuda_error();
  return(num_copied);
}

//------------------rednoise----------------//

template<typename T>
struct square {
    __host__ __device__ inline
    T operator()(const T& x) { return x*x; }
};

template<typename T>
float GPU_rms(T* d_collection,int nsamps, int min_bin)
{
  T rms_sum;
  float rms;

  using thrust::device_ptr;
  rms_sum = thrust::transform_reduce(device_ptr<T>(d_collection)+min_bin,
				     device_ptr<T>(d_collection)+nsamps,
				     square<T>(),T(0),thrust::plus<T>());
  rms = sqrt(float(rms_sum)/float(nsamps-min_bin));

  return rms;
}

template<typename T>
float GPU_mean(T* d_collection,int nsamps, int min_bin)
{
  float mean;
  T m_sum;

  using thrust::device_ptr;
  m_sum = thrust::reduce(device_ptr<T>(d_collection)+min_bin,
			 device_ptr<T>(d_collection)+nsamps);

  hipDeviceSynchronize();
  mean = float(m_sum)/float(nsamps-min_bin);

  return mean;
}

void device_normalise_spectrum(int nsamp,
      float* d_power_spectrum,
      float* d_normalised_power_spectrum,
      int min_bin,
      float * sigma)
{
  float mean;
  float rms;
  float meansquares;
  
  if (*sigma==0.0) {
    mean = GPU_mean(d_power_spectrum,nsamp,min_bin);
    rms = GPU_rms(d_power_spectrum,nsamp,min_bin);
    meansquares = rms*rms;
    *sigma = sqrt(meansquares - (mean*mean));
  }
  
  thrust::transform(thrust::device_ptr<float>(d_power_spectrum),
                    thrust::device_ptr<float>(d_power_spectrum)+nsamp,
                    thrust::make_constant_iterator(*sigma),
                    thrust::device_ptr<float>(d_normalised_power_spectrum),
                    thrust::divides<float>());
  ErrorChecker::check_cuda_error();
  
}

//--------------Time series folder----------------//


__global__ 
void rebin_time_series_kernel(float* i_data, float* o_data,
			      unsigned int size, float tsamp,
			      float period, unsigned int nbins,
			      unsigned int gulp_idx) 
{ 
  int ii;
  float val;
  int count;
  int idx = blockIdx.x * blockDim.x + threadIdx.x + gulp_idx;
  if (idx>size)
    return;
  int start_idx = __float2int_rn(idx*period/(tsamp*nbins));
  int end_idx = __float2int_rn((idx+1)*period/(tsamp*nbins));
  for (ii=start_idx;ii<end_idx;ii++)
    {
      val+=i_data[ii];
      count++;
    }
  o_data[idx] = val/count;
}


__global__ 
void create_subints_kernel(float* input, float* output,
			   unsigned int nbins,
			   unsigned int output_size,
			   unsigned int nrots_per_subint)
{
  int ii;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>output_size)
    return;
  unsigned int bin = idx%nbins;
  unsigned int subint = idx/nbins;
  unsigned int offset = subint*nrots_per_subint*nbins;
  float val = 0;
  for (ii=0;ii<nrots_per_subint;ii++)
    {
      val+=input[(ii*nbins)+bin+offset];
    }
  output[idx] = val/nrots_per_subint;
}

void device_create_subints(float* input, float* output,
			   unsigned int nbins,
                           unsigned int output_size,
                           unsigned int nrots_per_subint,
			   unsigned int max_blocks,
			   unsigned int max_threads)
{
  unsigned int nblocks = output_size/max_threads + 1;
  create_subints_kernel<<<nblocks,max_threads>>>(input,output,nbins,
						 output_size,
						 nrots_per_subint);
}


void device_rebin_time_series(float* input, float* output,
			      float period, float tsamp,
			      unsigned int in_size, unsigned int out_size,
			      unsigned int nbins,
			      unsigned int max_blocks, unsigned int max_threads)
{
  unsigned int gulps;
  unsigned int gulp_counter;
  unsigned int gulp_index = 0;
  unsigned int gulp_size;
  unsigned int blocks = 0;
  gulps = out_size/(max_blocks*max_threads)+1;
  for (gulp_counter = 0; gulp_counter<gulps; gulp_counter++)
    {
      if (gulp_counter<gulps-1)
	gulp_size = max_blocks*max_threads;
      else
	gulp_size = out_size-gulp_counter*max_blocks*max_threads;
      blocks = (gulp_size-1)/max_threads + 1;
      gulp_index += blocks*max_threads;
      rebin_time_series_kernel<<<blocks,max_threads>>>(input,output,out_size,
						       tsamp,period,nbins,
						       gulp_index);
    }
}

//--------------End--------------//
